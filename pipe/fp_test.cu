#include "hip/hip_runtime.h"

#include <stdio.h>
#include <time.h>
#define N (1024*1024)
#define M (1000)
#define THREADS_PER_BLOCK 1024
void serial_add(double *a, double *b, double *c, int n, int m)
{
	for(int index=0;index<n;index++)
	{
		for(int j=0;j<m;j++)
		{
			c[index] = a[index]*a[index] + b[index]*b[index];
		}
	}
}

__global__ void vector_add(double *a, double *b, double *c)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	for(int j=0;j<M;j++)
	{
		c[index]=a[index]*a[index]+b[index]*b[index];
	}
}

int main()
{
	clock_t start,end;
	double *a, *b, *c;
	int size=N*sizeof( double );
	a= (double *)malloc( size );
	b= (double *)malloc( size );
	c= (double *)malloc( size );
	for(int i=0;i<N;i++)
	{
		a[i]=b[i]=i;
		c[i]=0;
	}
	start=clock();
	serial_add(a,b,c,N,M);
	printf("c[%d]=%f\n",0,c[0]);
	printf("c[%d]=%f\n",N-1,c[N-1]);
	end=clock();
	float time1=((float)(end-start))/CLOCKS_PER_SEC;
	printf("CPU: %f seconds\n",time1);
	
	double *d_a,*d_b,*d_c;
	hipMalloc((void **) &d_a,size);
	hipMalloc((void **) &d_b,size);
	hipMalloc((void **) &d_c,size);
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
	start=clock();
	vector_add<<<(N+(THREADS_PER_BLOCK-1))/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a,d_b,d_c);
	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	printf("c[%d]=%f\n",0,c[0]);
	printf("c[%d]=%f\n",N-1,c[N-1]);
	end=clock();
	free(a);
	free(b);
	free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	
	float time2 = ((float)(end-start))/CLOCKS_PER_SEC;
	printf("CUDA: %f seconds, Speedup: %f\n",time2,time1/time2);
	return 0;
}