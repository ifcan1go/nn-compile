#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <fstream>
#include <sstream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <time.h>
#include <iostream>

using namespace std;

__global__ void CUDA_add(float *a,float *b,float *c,int cols,int rows)
{

        int n_cell= blockIdx.x ;
        int row  = blockIdx.y ;
        int col = threadIdx.x;
        c[n_cell*rows*cols+row*cols+col]=a[n_cell*rows*cols+row*cols+col]+b[n_cell*rows*cols+row*cols+col];

}
__global__ void CUDA_mul(float *a,float b,float *c,int cols,int rows)
{

        int n_cell= blockIdx.x ;
        int row  = blockIdx.y ;
        int col = threadIdx.x;
        c[n_cell*rows*cols+row*cols+col]=a[n_cell*rows*cols+row*cols+col]*b;

}


__global__ void CUDA_mmul(float *a,float *b,float *c,int cols,int rows)
{

        int n_cell= blockIdx.x ;
        int row  = blockIdx.y ;
        int col = threadIdx.x;
        c[n_cell*rows*cols+row*cols+col]=a[n_cell*rows*cols+row*cols+col]*b[n_cell*rows*cols+row*cols+col];
}

__global__ void CUDA_shift(float *a,float b,float *c,int cols,int rows)
{

        int n_cell= blockIdx.x ;
        int row  = blockIdx.y ;
        int col = threadIdx.x;
        c[n_cell*rows*cols+row*cols+col]=a[n_cell*rows*cols+row*cols+col]+b;
}
__global__ void CUDA_MatrixMui(float *a,float *b,float *c,int cols,int rows)
{
    int n_cell= blockIdx.x ;
    int row  = blockIdx.y ;
    //int col = threadIdx.x;
    float temp = 0;
    for (int i=0;i<cols;i++)
    {
        temp+=a[n_cell*cols+i]*b[n_cell*rows*cols+row*cols+i];

    }

    c[n_cell*rows+row]=temp;
}

#define DA_V 1.0f			//DA参考电压
#define AD_V 1.0f			//AD参考电压
#define DA_WIDTH 1			//DA输入数据宽度
#define AD_WIDTH 8			//AD输出数据宽度
#define CROSSBAR_L 1152		//crossbar长度
#define CROSSBAR_W 128		//crossbar宽度
#define CROSSBAR_N 1		//crossbar个数
#define AD_REUSE_NUM 32		//AD复用
#define XB01_I 0.00492679327726364
#define XB2_I 0.00398490577936172
#define XB3_I 0.00257207546383142

#define INPUT_SIZE 1152			//输入8bit数据个数

using namespace std;

typedef struct Crossbar1
{
	float *std_d;
	int CB_l;
	int CB_w;
	float *CB_cell;
	void init(float *CB_cells, int l, int w)
	{
		CB_l=l;
		CB_w=w;
		CB_cell = new float[CB_l*CB_w];
		memcpy(CB_cell, CB_cells, CB_l*CB_w * sizeof(float));
	}

	void MatrixMul(float *input, float *CB_cells, float *output, int w, int l)
	{
		for (int i = 0; i < w; i ++)
		{
			float tmp = 0;
			for (int j = 0; j < l; j ++)
			{
				tmp += input[j] * CB_cells[i*l+j];
			}
			output[i] = tmp;
			//cout << output[i] << endl;
		}
	}
	
    void run(float *input, float *output)
    {
		float *output_d = new float[CB_w];
		float *input_d = new float[CB_l];
		memcpy(input_d, input, CB_l*sizeof(float));
		MatrixMul(input_d,CB_cell,output_d,CB_w,CB_l);
		memcpy(output, output_d, CB_w* sizeof(float));
    }

}CROSSBAR1;

typedef struct Crossbar
{
	float *CB_cell;
	float *std_d;
	int CB_n;
	int CB_l;
	int CB_w;
    hiprandGenerator_t gen;
	void init(float *CB_cells, int n, int l, int w)
	{
		CB_l=l;
		CB_w=w;
		CB_n=n;
		hipMalloc((void **)&CB_cell, CB_n*CB_l*CB_w*sizeof(float));
		hipMemcpy(CB_cell, CB_cells, CB_n*CB_l*CB_w * sizeof(float),hipMemcpyHostToDevice);
		get_std();
		hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
		clock_t time;
	    time=clock();
	    hiprandSetPseudoRandomGeneratorSeed(gen, (int)time);

	}
	void printcrossbar()
	{
		float *temp_cell;
		temp_cell = (float*)calloc(CB_n*CB_l*CB_w,sizeof(float));
		//temp_cell = new float [CB_n*CB_l*CB_w];
		hipMemcpy(temp_cell, CB_cell, CB_n*CB_l*CB_w* sizeof(float),hipMemcpyDeviceToHost) ;
		printf ("_______________\n");
		for (int i=0;i<CB_n;i++)
		{
			for (int j=0;j<CB_l;j++)
			{
				for(int k=0;k<CB_w;k++)
				{
					printf("%f,%d,%d,%d,%d ",temp_cell[i*CB_l*CB_w+j*CB_w+k],i,j,k,i*CB_l*CB_w+j*CB_w+k);

				}
				printf ("\n");
			}
			printf ("\n");
		}
        printf ("_______________\n");
	free(temp_cell);
	}
	void get_std()
	//-0.0006034 * (x * 1e3) ** 2 + 0.06184 * x + 0.7240 * 1e-6
	{
		dim3 numBlocks(CB_n, CB_l);
		hipMalloc((void **)&std_d, CB_n*CB_l*CB_w*sizeof(float));
		float *temp_1;
		hipMalloc((void **)&temp_1, CB_n*CB_l*CB_w*sizeof(float));
		hipMemcpy(temp_1, CB_cell, CB_n*CB_l*CB_w* sizeof(float),hipMemcpyDeviceToDevice) ;
		float *temp_2;
		hipMalloc((void **)&temp_2, CB_n*CB_l*CB_w*sizeof(float));
		CUDA_mul<<<numBlocks,CB_l>>>(temp_1,1000,temp_2,CB_w,CB_l);
		hipMemcpy(temp_1, temp_2, CB_n*CB_l*CB_w* sizeof(float),hipMemcpyDeviceToDevice) ;
		float *temp_3;
		hipMalloc((void **)&temp_3, CB_n*CB_l*CB_w*sizeof(float));
		CUDA_mmul<<<numBlocks,CB_w>>>(temp_1,temp_2,temp_3,CB_w,CB_l);
		CUDA_mul<<<numBlocks,CB_w>>>(temp_3,-0.0006034,temp_1,CB_w,CB_l);
		CUDA_mul<<<numBlocks,CB_w>>>(CB_cell,0.06184,temp_2,CB_w,CB_l);
		CUDA_add<<<numBlocks,CB_w>>>(temp_1,temp_2,temp_3,CB_w,CB_l);
		CUDA_shift<<<numBlocks,CB_w>>>(temp_3,0.7240*0.000001,temp_1,CB_w,CB_l);
		hipMemcpy(std_d, temp_1, CB_n*CB_l*CB_w* sizeof(float),hipMemcpyDeviceToDevice) ;
		hipFree( temp_1 );
		hipFree( temp_2 );
		hipFree( temp_3 );
	}
	void get_noise(float *noise)
	{


	hipMalloc((void **)&noise, CB_n*CB_l*CB_w*sizeof(float));
	hiprandGenerateNormal(gen, noise, CB_n*CB_l*CB_w, 0, 1);
    //printf("%f\n", &noise[1]);
	}

    void printstd()
    {
            printf ("~~~~~~~~~~~~~~~~~~~\n");
            float *temp_cell;
            temp_cell = (float*)calloc(CB_n*CB_l*CB_w,sizeof(float));
            hipMemcpy(temp_cell, std_d, CB_n*CB_l*CB_w* sizeof(float),hipMemcpyDeviceToHost) ;
            for (int i=0;i<CB_n;i++)
            {
                    for (int j=0;j<CB_l;j++)
                    {
                            for(int p=0;p<CB_w;p++)
                            {
                                    printf("%f ",temp_cell[i*CB_l*CB_w+j*CB_w+p]);
                            }
                            printf ("\n");
                    }
                    printf ("\n");
            }
            printf ("~~~~~~~~~~~~~~~~~~~\n");
    free(temp_cell);
    }
    void run(float *input, float *output, bool use_noise=true)
    {
    
    float *input_d,*output_d;
    hipMalloc((void **)&input_d, CB_n*CB_w*sizeof(float));
    hipMalloc((void **)&output_d, CB_n*CB_l*sizeof(float));
    hipMemcpy(input_d, input, CB_n*CB_w * sizeof(float),hipMemcpyHostToDevice);
    dim3 numBlocks(CB_n, CB_l);
    if (use_noise)
        {
        float *temp_noise,*temp_cell;
        hipMalloc((void **)&temp_noise, CB_n*CB_w*CB_l*sizeof(float));
        hipMalloc((void **)&temp_cell, CB_n*CB_w*CB_l*sizeof(float));
        get_noise(temp_noise);
        //printf("%f\n", &temp_noise[1]);
        CUDA_add<<<numBlocks,CB_w>>>(CB_cell,temp_noise,temp_cell,CB_w,CB_l);
        CUDA_MatrixMui<<<numBlocks,1>>>(input_d,temp_cell,output_d,CB_w,CB_l);
        //printf("%f\n", &CB_cell[1109]);
        }
    else
        {
        //printf("%f\n", &CB_cell[5000]);
        CUDA_MatrixMui<<<numBlocks,1>>>(input_d,CB_cell,output_d,CB_w,CB_l);
        }
    hipMemcpy(output, output_d, CB_n*CB_l* sizeof(float),hipMemcpyDeviceToHost) ;
    hipFree( input_d );
	hipFree( output_d );
    }

}CROSSBAR;

double gaussrand()
{
    static double V1, V2, S;
    static int phase = 0;
    double X;
    if ( phase == 0 ) {
        do {
            double U1 = (double)rand() / RAND_MAX;
            double U2 = (double)rand() / RAND_MAX;
            V1 = 2 * U1 - 1;
            V2 = 2 * U2 - 1;
            S = V1 * V1 + V2 * V2;
        } while(S >= 1 || S == 0);
        X = V1 * sqrt(-2 * log(S) / S);
    } else
    X = V2 * sqrt(-2 * log(S) / S);
    phase = 1 - phase;
    return X;

}

float get_noise(float x)
{
    float noise;
    x = fabsf(x);
    float random = 
    noise = -0.0006034 * (x * 1000) * (x * 1000) + 0.06184 * x + 0.948661*0.000001;
    noise = noise * gaussrand();
    return noise;
}

int main()
{	
	float out[INPUT_SIZE];
	float da_res[CROSSBAR_L*(8/DA_WIDTH)];
	float out_data0[CROSSBAR_W*(8/DA_WIDTH)];
	float out_data1[CROSSBAR_W*(8/DA_WIDTH)];
	float res1[2*CROSSBAR_W];

	float da_res2[CROSSBAR_L*(8/DA_WIDTH)];
	float out_data2[CROSSBAR_W*(8/DA_WIDTH)];
	float res2[CROSSBAR_W];
	float da_res3[CROSSBAR_L*(8/DA_WIDTH)];
	float out_data3[CROSSBAR_W*(8/DA_WIDTH)];
	int res;

	int m = 0;//用作移位
	for (int i = 0; i < DA_WIDTH; i ++)
		m += int(pow(2, double(i)));

	CROSSBAR cb0;
	float* cell0 = new float[CROSSBAR_W*CROSSBAR_L];
	float** out_i_tmp0 = new float*[8/DA_WIDTH];
	for (int i = 0; i < 8/DA_WIDTH; i ++)
		out_i_tmp0[i] = new float[CROSSBAR_W];
	//权重读入
	ifstream inFile0("./Desktop/xb_0.csv", ios::in);
	string lineStr0;
	int C = 0;
	while (getline(inFile0, lineStr0))
	{
		//cout<<lineStr<<endl;
		stringstream ss(lineStr0);
		string str;
		int c = 0;
		while (getline(ss, str, ','))
		{
			istringstream iss(str);
			float num;
			iss >> num;
			cell0[C+CROSSBAR_L*c] = num + get_noise(num);
			c ++;
		}
		C ++;
	}

	CROSSBAR cb1;
	float* cell1 = new float[CROSSBAR_W*CROSSBAR_L];
	float** out_i_tmp1 = new float*[8/DA_WIDTH];
	for (int i = 0; i < 8/DA_WIDTH; i ++)
		out_i_tmp1[i] = new float[CROSSBAR_W];
	//权重读入
	ifstream inFile1("./Desktop/xb_1.csv", ios::in);
	string lineStr1;
	C = 0;
	while (getline(inFile1, lineStr1))
	{
		//cout<<lineStr<<endl;
		stringstream ss(lineStr1);
		string str;
		int c = 0;
		while (getline(ss, str, ','))
		{
			istringstream iss(str);
			float num;
			iss >> num;
			cell1[C+CROSSBAR_L*c] = num + get_noise(num);
			c ++;
		}
		C ++;
	}

	CROSSBAR cb2;
	float* cell2 = new float[CROSSBAR_W*CROSSBAR_L];
 	float** out_i_tmp2 = new float*[8/DA_WIDTH];
	for (int i = 0; i < 8/DA_WIDTH; i ++)
		out_i_tmp2[i] = new float[CROSSBAR_W];
	//权重读入
	ifstream inFile2("./Desktop/xb_2.csv", ios::in);
	string lineStr2;
	C = 0;
	while (getline(inFile2, lineStr2))
	{
		//cout<<lineStr<<endl;
		stringstream ss(lineStr2);
		string str;
		int c = 0;
		while (getline(ss, str, ','))
		{
			istringstream iss(str);
			float num;
			iss >> num;
			cell2[C+CROSSBAR_L*c] = num + get_noise(num);
			c ++;
		}
		C ++;
	}

	CROSSBAR cb3;
	float* cell3 = new float[CROSSBAR_W*CROSSBAR_L];
	float** out_i_tmp3 = new float*[8/DA_WIDTH];
	for (int i = 0; i < 8/DA_WIDTH; i ++)
		out_i_tmp3[i] = new float[CROSSBAR_W];
	//权重读入
	ifstream inFile3("./Desktop/xb_3.csv", ios::in);
	string lineStr3;
	C = 0;
	while (getline(inFile3, lineStr3))
	{
		//cout<<lineStr<<endl;
		stringstream ss(lineStr3);
		string str;
		int c = 0;
		while (getline(ss, str, ','))
		{
			istringstream iss(str);
			float num;
			iss >> num;
			cell3[C+CROSSBAR_L*c] = num + get_noise(num);
			c ++;
		}
		C ++;
	}
	
	


	for (int k = 9774; k < 10000; k ++)
	{
		char filename[30]={0};
		char num[5]={0};
		strcpy(filename,"./x/");
		itoa(k,num,10);
		strcat(filename,num);
		strcat(filename,".csv");
		ifstream inFile_x(filename, ios::in);
		string lineStr_x;
		getline(inFile_x, lineStr_x);
		stringstream ss(lineStr_x);
		string str;
		int c = 0;
		for (int i = 0; i < 368; i ++)
			out[i] = 0;
		while (getline(ss, str, ','))
		{
			istringstream iss(str);
			float num;
			iss >> num;
			out[368+c] = num;
			c ++;
		}
		inFile_x.close();
		//cout << "1: " << out[1108] << endl;
		int* data0 = new int[INPUT_SIZE];
		for (int i = 0; i < INPUT_SIZE; i ++)
			data0[i] = int(out[i]);
		int bitnum;
		for (int j = 8/DA_WIDTH-1; j >= 0; j--)
		{
			for (int i = 0; i < INPUT_SIZE; i ++)
			{
				bitnum = static_cast<int>(data0[i] & m);
				da_res[i+j*INPUT_SIZE] = bitnum;
				data0[i] >>= DA_WIDTH;
			}
		}
		//cout << "2: " << da_res[1108+1152*2] << endl;
		float** tmp_v = new float*[8/DA_WIDTH];
		for (int i = 0; i < 8/DA_WIDTH; i ++)
			tmp_v[i] = new float[CROSSBAR_L];
		for (int i = 0; i < 8/DA_WIDTH; i ++)
		{
			for (int j = 0; j < CROSSBAR_L; j ++)
				tmp_v[i][j] = da_res[i*CROSSBAR_L+j];
		}
		//cout << "3: " << tmp_v[0][1108] << endl;
		
		//cout << cell0[368] << endl;
		cb0.init(cell0, 1, CROSSBAR_W, CROSSBAR_L);
		
		for (int i = 0; i < 8/DA_WIDTH; i ++)
		{
			float* v_in = new float[CROSSBAR_L];
			float* i_out = new float[CROSSBAR_W];
			for (int j = 0; j < CROSSBAR_L; j ++)
			{
				v_in[j] = tmp_v[i][j];
			}
			cb0.run(v_in, i_out, false);
			for (int j = 0; j < CROSSBAR_W; j ++)
				out_i_tmp0[i][j] = i_out[j];
		}
		//cout << out_i_tmp0[0][0] << endl;
		
		//cout << cell1[368] << endl;
		cb1.init(cell1, 1, CROSSBAR_W, CROSSBAR_L);
		for (int i = 0; i < 8/DA_WIDTH; i ++)
		{
			cb1.run(tmp_v[i], out_i_tmp1[i], false);
		}
		//cout << out_i_tmp1[0][1] << endl;

		float max_i01 = 0;
		for (int i = 0; i < 8/DA_WIDTH; i ++)
		{
			for (int j = 0; j < CROSSBAR_W; j ++)
			{
				if (out_i_tmp0[i][j] > max_i01)
					max_i01 = out_i_tmp0[i][j];
				if (out_i_tmp1[i][j] > max_i01)
					max_i01 = out_i_tmp1[i][j];
			}
		}
		//cout << max_i01 << endl;
		float* tmp_ad0 = new float[CROSSBAR_W*8/DA_WIDTH];//记录ad输出
		float* tmp_ad1 = new float[CROSSBAR_W*8/DA_WIDTH];
		for (int i = 0; i < 8/DA_WIDTH; i ++)
		{
			for (int j = 0; j < CROSSBAR_W; j ++)
			{
				tmp_ad0[j+i*CROSSBAR_W] = out_i_tmp0[i][j] * 255 / max_i01;
				tmp_ad1[j+i*CROSSBAR_W] = out_i_tmp1[i][j] * 255 / max_i01;

				tmp_ad0[j+i*CROSSBAR_W] = (tmp_ad0[j+i*CROSSBAR_W] > 0)?floor(tmp_ad0[j+i*CROSSBAR_W] + 0.5):ceil(tmp_ad0[j+i*CROSSBAR_W] - 0.5);
				tmp_ad1[j+i*CROSSBAR_W] = (tmp_ad1[j+i*CROSSBAR_W] > 0)?floor(tmp_ad1[j+i*CROSSBAR_W] + 0.5):ceil(tmp_ad1[j+i*CROSSBAR_W] - 0.5);

				out_data0[j+i*CROSSBAR_W] = tmp_ad0[j+i*CROSSBAR_W];
				out_data1[j+i*CROSSBAR_W] = tmp_ad1[j+i*CROSSBAR_W];
				//cout << tmp[i] << endl;
			}
		}
		//cout << out_data1[0] << endl;
		float** tmp0 = new float*[8/DA_WIDTH];
		float** tmp1 = new float*[8/DA_WIDTH];
		for (int i = 0; i < 8/DA_WIDTH; i ++)
		{
			tmp0[i] = new float[CROSSBAR_W];
			tmp1[i] = new float[CROSSBAR_W];
		}
		for (int i = 0; i < 8/DA_WIDTH; i ++)
		{
			for (int j = 0; j < CROSSBAR_W; j ++)
			{
				tmp0[i][j] = out_data0[i*CROSSBAR_W+j];
				tmp1[i][j] = out_data1[i*CROSSBAR_W+j];
			}
		}
		//cout << "stage3 input test: " << tmp1[3][127] << endl;

		float* tmp_res0 = new float[CROSSBAR_W];
		float* tmp_res1 = new float[CROSSBAR_W];
		for (int i = 0; i < CROSSBAR_W; i ++)
		{
			tmp_res0[i] = 0;
			tmp_res1[i] = 0;
		}
		for (int i = 0; i < CROSSBAR_W; i ++)
		{
			//wait(10, SC_NS);
			for (int j = 0; j < 8/DA_WIDTH; j ++)
			{
				tmp_res0[i] = tmp0[j][i] + 2*tmp_res0[i];
				tmp_res1[i] = tmp1[j][i] + 2*tmp_res1[i];
			}
			tmp_res0[i] = (tmp_res0[i] > 0) ? tmp_res0[i] : 0;
			tmp_res1[i] = (tmp_res1[i] > 0) ? tmp_res1[i] : 0;
			//cout << tmp_res1[i] << endl;

			res1[i] = tmp_res0[i];
			res1[i+CROSSBAR_W] = tmp_res1[i];
		}
		//cout << res1[255] << endl;

		int* data = new int[CROSSBAR_L];
		for (int i = 0; i < CROSSBAR_L-2*CROSSBAR_W; i ++)
			data[i] = 0;
		for (int i = 0; i < 2*CROSSBAR_W; i ++)
		{
			data[CROSSBAR_L-2*CROSSBAR_W+i] = int(res1[i]);
			//cout << data[1052+i] << endl;
		}
		//cout << data[1151] << endl;
		int high = 0;
		int max = -1;
		int max_index = 0;
		for (int i = 0; i < CROSSBAR_L; i ++)
		{
			if (data[i] > max)
			{
				max = data[i];
				max_index = i;
			}
		}
		for (int i = 31; i >= 0; i --)
		{
			int m = (data[max_index] >> i) & 1;
			if (m == 1)
			{
				high = i+1;
				break;
			}
		}
		//cout << high << endl;
		for (int j = 8/DA_WIDTH-1; j >= 0; j--)
		{
			int move = DA_WIDTH*(8/DA_WIDTH-1-j);
			int bitnum;
			for (int i = 0; i < INPUT_SIZE; i ++)
			{
				bitnum = static_cast<int>((data[i] >> (high - 8 + move)) & m);
				da_res2[i+j*INPUT_SIZE] = bitnum;
			}
		}

		//cout << da_res2[1152*3-1] << endl;
		float** tmp_v2 = new float*[8/DA_WIDTH];
		for (int i = 0; i < 8/DA_WIDTH; i ++)
			tmp_v2[i] = new float[CROSSBAR_L];
		for (int i = 0; i < 8/DA_WIDTH; i ++)
		{
			for (int j = 0; j < CROSSBAR_L; j ++)
				tmp_v2[i][j] = da_res2[i*CROSSBAR_L+j];
		}
		//cout << tmp_v2[7][1151] << endl;
		
		//cout << cell2[1151] << endl;
		cb2.init(cell2, 1, CROSSBAR_W, CROSSBAR_L);
		for (int i = 0; i < 8/DA_WIDTH; i ++)
		{
			cb2.run(tmp_v2[i], out_i_tmp2[i], false);
		}

		float max_i2 = 0;
		for (int i = 0; i < 8/DA_WIDTH; i ++)
		{
			for (int j = 0; j < CROSSBAR_W; j ++)
			{
				if (out_i_tmp2[i][j] > max_i2)
					max_i2 = out_i_tmp2[i][j];
			}
		}
		
		float* tmp_ad2 = new float[CROSSBAR_W*8/DA_WIDTH];//记录ad输出
		for (int i = 0; i < 8/DA_WIDTH; i ++)
		{
			for (int j = 0; j < CROSSBAR_W; j ++)
			{
				tmp_ad2[j+i*CROSSBAR_W] = out_i_tmp2[i][j] * 255 / max_i2;
				tmp_ad2[j+i*CROSSBAR_W] = (tmp_ad2[j+i*CROSSBAR_W] > 0)?floor(tmp_ad2[j+i*CROSSBAR_W] + 0.5):ceil(tmp_ad2[j+i*CROSSBAR_W] - 0.5);
				out_data2[j+i*CROSSBAR_W] = tmp_ad2[j+i*CROSSBAR_W];
				//cout << tmp[i] << endl;
			}
		}

		float** tmp2 = new float*[8/DA_WIDTH];
		for (int i = 0; i < 8/DA_WIDTH; i ++)
			tmp2[i] = new float[CROSSBAR_W];
		for (int i = 0; i < 8/DA_WIDTH; i ++)
		{
			for (int j = 0; j < CROSSBAR_W; j ++)
			{
				tmp2[i][j] = out_data2[i*CROSSBAR_W+j];
			}
		}
		//cout << "stage6 input test: " << tmp[0][1] << endl;
		float* tmp_res2 = new float[CROSSBAR_W];
		for (int i = 0; i < CROSSBAR_W; i ++)
			tmp_res2[i] = 0;
		for (int i = 0; i < CROSSBAR_W; i ++)
		{
			//wait(10, SC_NS);
			for (int j = 0; j < 8/DA_WIDTH; j ++)
			{
				tmp_res2[i] = tmp2[j][i] + 2*tmp_res2[i];
			}
			tmp_res2[i] = (tmp_res2[i] > 0) ? tmp_res2[i] : 0;
			res2[i] = tmp_res2[i];
		}
		//cout << res2[127] << endl;
		int* data3 = new int[CROSSBAR_L];
		for (int i = 0; i < CROSSBAR_L-CROSSBAR_W; i ++)
			data3[i] = 0;
		for (int i = 0; i < CROSSBAR_W; i ++)
		{
			data3[CROSSBAR_L-CROSSBAR_W+i] = int(res2[i]);
			//cout << data[1052+i] << endl;
		}
		//cout << data3[1151] << endl;
		int high3 = 0;
		int max3 = -1;
		int max_index3 = 0;
		for (int i = 0; i < CROSSBAR_L; i ++)
		{
			if (data3[i] > max3)
			{
				max3 = data3[i];
				max_index3 = i;
			}
		}
		//cout << max3 << endl;
		for (int i = 31; i >= 0; i --)
		{
			int m = (data3[max_index3] >> i) & 1;
			if (m == 1)
			{
				high3 = i+1;
				break;
			}
		}
		//cout << high3 << endl;
		for (int j = 8/DA_WIDTH-1; j >= 0; j--)
		{
			int move = DA_WIDTH*(8/DA_WIDTH-1-j);
			int bitnum;
			for (int i = 0; i < INPUT_SIZE; i ++)
			{
				bitnum = static_cast<int>((data3[i] >> (high3 - 8 + move)) & m);
				da_res3[i+j*INPUT_SIZE] = bitnum;
			}
		}

		float** tmp_v3 = new float*[8/DA_WIDTH];
		for (int i = 0; i < 8/DA_WIDTH; i ++)
			tmp_v3[i] = new float[CROSSBAR_L];
		for (int i = 0; i < 8/DA_WIDTH; i ++)
		{
			for (int j = 0; j < CROSSBAR_L; j ++)
				tmp_v3[i][j] = da_res3[i*CROSSBAR_L+j];
		}
		
		cb3.init(cell3, 1, CROSSBAR_W, CROSSBAR_L);
		for (int i = 0; i < 8/DA_WIDTH; i ++)
		{
			cb3.run(tmp_v3[i], out_i_tmp3[i], false);
		}

		float max_i3 = 0;
		for (int i = 0; i < 8/DA_WIDTH; i ++)
		{
			for (int j = 0; j < CROSSBAR_W; j ++)
			{
				if (out_i_tmp3[i][j] > max_i3)
					max_i3 = out_i_tmp0[i][j];
			}
		}
		
		float* tmp_ad3 = new float[CROSSBAR_W*8/DA_WIDTH];//记录ad输出
		for (int i = 0; i < 8/DA_WIDTH; i ++)
		{
			for (int j = 0; j < CROSSBAR_W; j ++)
			{
				tmp_ad3[j+i*CROSSBAR_W] = out_i_tmp3[i][j] * 255 / max_i3;
				tmp_ad3[j+i*CROSSBAR_W] = (tmp_ad3[j+i*CROSSBAR_W] > 0)?floor(tmp_ad3[j+i*CROSSBAR_W] + 0.5):ceil(tmp_ad3[j+i*CROSSBAR_W] - 0.5);
				out_data3[j+i*CROSSBAR_W] = tmp_ad3[j+i*CROSSBAR_W];
				//cout << tmp[i] << endl;
			}
		}

		float** tmp3 = new float*[8/DA_WIDTH];
		for (int i = 0; i < 8/DA_WIDTH; i ++)
			tmp3[i] = new float[CROSSBAR_W];
		for (int i = 0; i < 8/DA_WIDTH; i ++)
		{
			for (int j = 0; j < CROSSBAR_W; j ++)
			{
				tmp3[i][j] = out_data3[i*CROSSBAR_W+j];
			}
		}
		//cout << "stage6 input test: " << tmp[0][1] << endl;
		float* tmp_res3 = new float[CROSSBAR_W];
		for (int i = 0; i < CROSSBAR_W; i ++)
			tmp_res3[i] = 0;
		for (int i = 0; i < CROSSBAR_W; i ++)
		{
			//wait(10, SC_NS);
			for (int j = 0; j < 8/DA_WIDTH; j ++)
			{
				tmp_res3[i] = tmp3[j][i] + 2*tmp_res3[i];
			}
			tmp_res3[i] = (tmp_res3[i] > 0) ? tmp_res3[i] : 0;
		}

		float max4 = 0;
		int index4 = 0;
		for (int i = 0; i < 10; i ++)
		{
			if (tmp_res3[i] > max4)
			{
				max4 = tmp_res3[i];
				index4 = i;
			}
		}

		res = index4;
		cout << "RES: " << res << endl;

		delete[] data0;
		delete[] data;
		delete[] data3;
		for (int i = 0; i < 8/DA_WIDTH; i ++)
		{
			delete[] tmp_v[i];
		}
		delete[] tmp_v;
		delete[] tmp_ad0;
		delete[] tmp_ad1;
		delete[] tmp_ad2;
		delete[] tmp_ad3;

		for (int i = 0; i < 8/DA_WIDTH; i ++)
		{
			delete[] tmp0[i];
			delete[] tmp1[i];
			delete[] tmp2[i];
			delete[] tmp3[i];
		}
		delete[] tmp0;
		delete[] tmp1;
		delete[] tmp2;
		delete[] tmp3;
		delete[] tmp_res0;
		delete[] tmp_res1;
		delete[] tmp_res2;
		delete[] tmp_res3;

	}
	return 0;
}